#include "hip/hip_runtime.h"

/// Kernal finds smallest value in input vector A, by using reduction pattern to compare two values and saving smallest.
/// This repeats until the total smallest value repeating until smallest value in data is stored in the first index of B
__global__ void reduce_find_min(const int* A, int* B, __shared__ int* scratch) 
{
	int X = blockIdx.x * blockDim.x + threadIdx.x;
	int Y = threadIdx;
	int Z = blockDim.x;
	scratch[Y] = A[X];
	__syncthreads();
	for (int i = 1; i < Z; i *= 2) 
	{
		if (!(Y % (i * 2)) && ((Y + i) < Z)) 
			scratch[Y] = (scratch[Y] < scratch[Y + i]) ? scratch[Y] : scratch[Y + i];
		__syncthreads();
	}
	if (!Y) 
	{
		atomicMin(&B[0],scratch[Y]);
	}
}

/// Same kernal as the first one but reversed
__global__ void reduce_find_max(__device__ const int* A, __device__ int* B, __shared__ int* scratch) 
{
	int X = blockIdx.x * blockDim.x + threadIdx.x;
	int Y = threadIdx;
	int Z = blockDim.x;
	scratch[Y] = A[X];
	__syncthreads();
	for (int i = 1; i < Z; i *= 2) 
	{
		if (!(Y % (i * 2)) && ((Y + i) < Z)) 
			scratch[Y] = (scratch[Y] > scratch[Y + i]) ? scratch[Y] : scratch[Y + i];
		__syncthreads();
	}
	if (!Y) 
	{
		atomicMax(&B[0],scratch[Y]);
	}
}

__global__ void reduce_find_sum(__device__ const int* A, __device__ int* B, __shared__ int* scratch) 
{
	int X = blockIdx.x * blockDim.x + threadIdx.x;
	int Y = threadIdx;
	int Z = blockDim.x;
	scratch[Y] = A[X];
	__syncthreads();
	for (int i = 1; i < Z; i *= 2) 
	{
		if (!(Y % (i * 2)) && ((Y + i) < Z)) 
			scratch[Y] += scratch[Y + i];
		__syncthreads();
	}
	if (!Y) 
	{
		atomic_add(&B[0],scratch[Y]);
	}
}

__global__ void reduce_find_sum_variance(__device__ const int* A, __device__ int* B, __shared__ int* scratch) 
{
	int X = blockIdx.x * blockDim.x + threadIdx.x;
	int Y = threadIdx;
	int Z = blockDim.x;
	scratch[Y] = A[X];
	__syncthreads();
	for (int i = 1; i < Z; i *= 2) 
	{
		if (!(Y % (i * 2)) && ((Y + i) < Z)) 
			scratch[Y] += scratch[Y + i];
		__syncthreads();
	}
	scratch[Y] = scratch[Y] / 10000.0f;
	if (!Y) 
	{
		atomic_add(&B[0],scratch[Y]);
	}
}

__global__ void find_variance(__device__ const int* A, __device__ int* B, int mean, int initialSize) 
{
	int X = blockIdx.x * blockDim.x + threadIdx.x;
	if(X < initialSize)
	{ 
		B[X] = A[X] - mean;
		__syncthreads();
		B[X] = (B[X] * B[X]);
	}
}

__global__ void at_find_min(__device__ const int* A, __device__ int* B, __shared__ int* scratch) 
{
	int X = blockIdx.x * blockDim.x + threadIdx.x;
	int Y = threadIdx;
	scratch[Y] = A[X];
	__syncthreads();
	atomicMin(&B[0],scratch[Y]);
}

__global__ void at_find_max(__device__ const int* A, __device__ int* B, __shared__ int* scratch) 
{
	int X = blockIdx.x * blockDim.x + threadIdx.x;
	int Y = threadIdx;
	scratch[Y] = A[X];
	__syncthreads(); 
	atomicMax(&B[0],scratch[Y]);
}